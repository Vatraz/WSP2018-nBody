#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <>
#include <windows.h>
#include <helper_gl.h>
#include <GL/freeglut.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <helper_functions.h>    
#include <timer.h>      
#include <chrono>
#include <hip/hip_runtime_api.h>         


#define REFRESH_DELAY		10 //ms
#define ZMIEKCZANIE			0.001f
#define G					0.66742f
#define KROK				0.005f

using namespace std;
using namespace std::chrono;


// Wymiary okna
const int okno_szerokosc = 800;
const int okno_wysokosc = 500;

int liczba_cial;
const int liczba_watkow = 256;
int liczba_blokow;
float czas_symulacji = 0.0;
float mnoznik_predkosci;

// OpenGL Vertex Buffer Object
GLuint vbo;

// Inicjalizacja obiektow/ buforow obiektow
float4 *d_obiekty_v;
float4 *h_obiekty_v;
float4 *h_obiekty_wsp_m;

// licznik FPS 
high_resolution_clock::time_point timer1;
high_resolution_clock::time_point timer2;
int frameCount = 0;
float avg_FPS = 0.0f;
int limit_FPS = 100;  

// prototypy funkcji
bool initGL(int *argc, char **argv);
void stworz_tablice();
void stworz_VBO(GLuint *vbo);
void uruchomienie_kernela();
void losowanie(float4 *obiekty_v, float4 *obiekty_wsp_m, int n);
void cpu_test();
void cleanup();

// callbacks
void display();
void keyboard(unsigned char key, int x, int y);
void mouse(int button, int state, int x, int y);
void motion(int x, int y);
void timerEvent(int value);

int mouse_old_x, mouse_old_y;
int mouse_buttons = 0;
float rotate_x = 0.0;
float rotate_y = 0.0;
float translate_z = -20.0;

// wyliczenie sily wypadkowej dzialajacej na cialo
__device__ float3 sily(float4 *obiekty_wsp_m, float4 *obiekty_v, float4 ten_obiekt, int liczba_cial) {
	
	float3 r;
	float3 a = { 0.0f, 0.0f, 0.0f };
	float kwadrat_wektorow;
	float mianownik;

	for (int i = 0; i < liczba_cial; i++) {
		// aktualna pozycja
		r.x = obiekty_wsp_m[i].x - ten_obiekt.x;
		r.y = obiekty_wsp_m[i].y - ten_obiekt.y;
		r.z = obiekty_wsp_m[i].z - ten_obiekt.z;

		// liczenie odleglosci
		kwadrat_wektorow = r.x*r.x + r.y*r.y + r.z*r.z + ZMIEKCZANIE;
		mianownik = 1.f / kwadrat_wektorow;
		
		// obliczanie przyspieszen - kieunku wektora
		float s = G * ten_obiekt.w * mianownik;
		a.x += r.x * s;
		a.y += r.y * s;
		a.z += r.z * s;
	}
	return a;
}

// wyliczenie pozycji ciala
__global__ void pozycje(float4 *obiekty_wsp_m, float4 *obiekty_v, int liczba_cial, float mnoznik_predkosci)
{	
	__shared__  float4 obiekty_wsp_SHARED[liczba_watkow];
	int numer_obiektu = blockIdx.x * blockDim.x + threadIdx.x;
	if (numer_obiektu < liczba_cial) {
		obiekty_wsp_SHARED[threadIdx.x] = obiekty_wsp_m[numer_obiektu];

		__syncthreads();

		// liczenie przyspieszen
		float3 a;
		a = sily(obiekty_wsp_m, obiekty_v, obiekty_wsp_SHARED[threadIdx.x], liczba_cial);
		obiekty_v[numer_obiektu].x += a.x * KROK * mnoznik_predkosci;
		obiekty_v[numer_obiektu].y += a.y * KROK * mnoznik_predkosci;
		obiekty_v[numer_obiektu].z += a.z * KROK * mnoznik_predkosci;
									
		__syncthreads();

		// nowe pozycje obiektow
		obiekty_wsp_SHARED[threadIdx.x].x += obiekty_v[numer_obiektu].x * KROK;
		obiekty_wsp_SHARED[threadIdx.x].y += obiekty_v[numer_obiektu].y * KROK;
		obiekty_wsp_SHARED[threadIdx.x].z += obiekty_v[numer_obiektu].z * KROK;

		__syncthreads();

		// wpisanie nowej pozycji
		obiekty_wsp_m[numer_obiektu] = obiekty_wsp_SHARED[threadIdx.x];
	}
}

int main(int argc, char **argv)
{
	printf("Liczba cial: ");
	scanf("%i", &liczba_cial);
	printf("Mnoznik predkosci: ");
	scanf("%f", &mnoznik_predkosci);
	liczba_blokow = ((liczba_cial-1) / liczba_watkow) + 1;

	// openGL init
	initGL(&argc, argv);

	// stworzenie VBO- empty vertex buffer object, inicjalizacja 
	stworz_tablice();
	stworz_VBO(&vbo);
	
	// register callbacks
	glutDisplayFunc(display);
	glutKeyboardFunc(keyboard);
	glutMouseFunc(mouse);
	glutMotionFunc(motion);
	glutCloseFunc(cleanup);

	// cuda
	high_resolution_clock::time_point gpu1 = high_resolution_clock::now();
	uruchomienie_kernela();
	hipDeviceSynchronize();
	high_resolution_clock::time_point gpu2 = high_resolution_clock::now();
	duration<double> gput = duration_cast<duration<double>>(gpu2 - gpu1);

	// cpu
	high_resolution_clock::time_point cpu1 = high_resolution_clock::now();
	cpu_test();
	high_resolution_clock::time_point cpu2 = high_resolution_clock::now();
	duration<double> cput = duration_cast<duration<double>>(cpu2 - cpu1);

	// porownanie czasow
	printf("\nCzas potrzebny na wyliczenie jednego kroku symulacji: ");
	printf("\nCPU: %E", cput.count());
	printf("\nGPU: %E\n", gput.count());
	printf("\nPrzyspieszenie x%f\n\n", cput.count()/ gput.count());

	// glowna petla
	glutMainLoop();
	return 0;
}

// wykonanie obliczen na gpu
void uruchomienie_kernela()
{
	// mapuje bufor OpenGL do CUDA
	float4 *dptr;
	cudaGLMapBufferObject((void**)&dptr, vbo);

	pozycje << <liczba_blokow, liczba_watkow >> >(dptr, d_obiekty_v, liczba_cial, mnoznik_predkosci);

	// unmap vpo
	cudaGLUnmapBufferObject(vbo);
}

// Petla animacji
void display()
{	
	// uruchomienie licznika FPS
	if (frameCount == 0 )
		timer1 = high_resolution_clock::now();

	// wykonanie obliczen na GPU
	uruchomienie_kernela();
	
	// ustawienie macierzy widoku
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();
	glTranslatef(0.0, 0, translate_z);
	glRotatef(rotate_x, 1.0, 0.0, 0.0);
	glRotatef(rotate_y, 0.0, 1.0, 0.0);

	// renderowanie 
	glBindBuffer(GL_ARRAY_BUFFER, vbo);
	glVertexPointer(4, GL_FLOAT, 0, 0);
	glEnableClientState(GL_VERTEX_ARRAY);
	glColor3f(0.5, 0.5, 1.0);
	glDrawArrays(GL_POINTS, 0, liczba_cial);
	glDisableClientState(GL_VERTEX_ARRAY);
	glutSwapBuffers();

	// obsluga licznika FPS
	czas_symulacji += KROK;
	frameCount++;
	if (frameCount >= limit_FPS)
	{
		timer2 = high_resolution_clock::now();
		duration<double> czasFps = duration_cast<duration<double>>(timer2 - timer1);
		avg_FPS = frameCount / czasFps.count() ;
		frameCount = 0;
		char fps[256];
		sprintf(fps, "nBody: %2.2f fps", avg_FPS);
		glutSetWindowTitle(fps);
	}
	
}

// inicjalizacja openGL
bool initGL(int *argc, char **argv)
{
	glutInit(argc, argv);
	glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
	glutInitWindowSize(okno_szerokosc, okno_wysokosc);
	glutCreateWindow("nBody:");
	glutDisplayFunc(display);
	glutKeyboardFunc(keyboard);
	glutMotionFunc(motion);
	glutTimerFunc(REFRESH_DELAY, timerEvent, 0);

	if (!isGLVersionSupported(2, 0))
	{
		fprintf(stderr, "ERROR: Support for necessary OpenGL extensions missing.");
		fflush(stderr);
		return false;
	}

	glClearColor(0.0, 0.0, 0.0, 1.0);
	glDisable(GL_DEPTH_TEST);

	// viewport
	glViewport(0, 0, okno_szerokosc, okno_wysokosc);

	// projection
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	gluPerspective(60.0, (GLfloat)okno_szerokosc / (GLfloat)okno_wysokosc, 0.1, 100.0);

	SDK_CHECK_ERROR_GL();

	return true;
}

// tablice zawierajace polozenie i predksoc cial
void stworz_tablice() 
{
	int rozmiar_obiektow = 4 * liczba_cial * sizeof(float);

	// alokancja elegancka
	h_obiekty_v = (float4*)malloc(rozmiar_obiektow);
	h_obiekty_wsp_m = (float4*)malloc(rozmiar_obiektow);
	hipMalloc(&d_obiekty_v, rozmiar_obiektow);

	// zerowanie predkosci, losowanie polozenia i masy
	losowanie(h_obiekty_v, h_obiekty_wsp_m, liczba_cial);

	// kopiowanie wylosowanych danych do device
	hipMemcpy(d_obiekty_v, h_obiekty_v, rozmiar_obiektow, hipMemcpyHostToDevice);
}

// tworzenie OpenGL Vertex Buffer Object
void stworz_VBO(GLuint *vbo)
{
	int rozmiar_obiektow = 4 * liczba_cial * sizeof(float);
	
	assert(vbo);

	// tworzenie bufora
	glGenBuffers(1, vbo);
	glBindBuffer(GL_ARRAY_BUFFER, *vbo);

	glBufferData(GL_ARRAY_BUFFER, rozmiar_obiektow, h_obiekty_wsp_m, GL_DYNAMIC_DRAW);

	glBindBuffer(GL_ARRAY_BUFFER, 0);

	checkCudaErrors(cudaGLRegisterBufferObject(*vbo));

	SDK_CHECK_ERROR_GL();
}

// w ktorej czesci znajduja sie ciala
int znak() {
	int plusminus = rand() & 5;

	if (plusminus > 2)
		plusminus = 1;
	else
		plusminus = -1;
	return plusminus;
}

// losowanie polozen i predkosci cial
void losowanie(float4 *obiekty_v, float4 *obiekty_wsp_m, int n)
{
	float x; //e10
	float y; //e10
	float z; //e10

	for (int i = 0; i < n; i++) {
		x = znak() * 5.f * (rand() / (float)RAND_MAX); //e10
		y = znak() * 5.f * (rand() / (float)RAND_MAX); //e10
		z = znak() * 5.f * (rand() / (float)RAND_MAX); //e10
		obiekty_v[i] = make_float4(0.f, 0.f, 0.f, 1.f);
		obiekty_wsp_m[i] = make_float4(x, y, z, 2.f);
	}
}

// sily liczone na CPU
float3 CPUsily(float4 *obiekty_wsp_m, int indeks)
{
	float3 r;
	float3 a = { 0.0f, 0.0f, 0.0f };
	float kwadrat_wektorow;
	float distSqr3;
	float mianownik;

	for (int i = 0; i < liczba_cial; i++) {
		// aktualna pozycja
		r.x = obiekty_wsp_m[i].x - obiekty_wsp_m[indeks].x;
		r.y = obiekty_wsp_m[i].y - obiekty_wsp_m[indeks].y;
		r.z = obiekty_wsp_m[i].z - obiekty_wsp_m[indeks].z;

		// liczenie odleglosci
		kwadrat_wektorow = r.x*r.x + r.y*r.y + r.z*r.z + ZMIEKCZANIE;
		distSqr3 = kwadrat_wektorow;
		mianownik = 1.f / distSqr3;

		// obliczanie przyspieszen - kieunku wektora

		float s = G * obiekty_wsp_m[i].w * mianownik;
		a.x += r.x * s;
		a.y += r.y * s;
		a.z += r.z * s;
	}
	return a;
}

// pozycje cial liczone na CPU
void CPUpozycje(float4 *obiekty_wsp_m, float4 *obiekty_v)
{
	// liczenie przyspieszen
	float3 a;
	for (int i = 0; i < liczba_cial; i++) {
		a = CPUsily(obiekty_wsp_m, i);
		obiekty_v[i].x += a.x * KROK;
		obiekty_v[i].y += a.y * KROK;
		obiekty_v[i].z += a.z * KROK;
	}

	// nowe pozycje obiektow
	for (int i = 0; i < liczba_cial; i++) {
		obiekty_wsp_m[i].x += obiekty_v[i].x * KROK;
		obiekty_wsp_m[i].y += obiekty_v[i].y * KROK;
		obiekty_wsp_m[i].z += obiekty_v[i].z * KROK;
	}
}

// test wydajnosci CPU
void cpu_test() {
	CPUpozycje(h_obiekty_wsp_m, h_obiekty_v);
}

// usuniecie vbo
void deleteVBO(GLuint *vbo)
{
	glBindBuffer(1, *vbo);
	glDeleteBuffers(1, vbo);
	*vbo = 0;
}

// czyszczenie po zamknieciu
void cleanup()
{
	deleteVBO(&vbo);
	hipFree(d_obiekty_v);
	free(h_obiekty_v);
	free(h_obiekty_wsp_m);
}

// obsluga klawiatury
void keyboard(unsigned char key, int /*x*/, int /*y*/)
{
	switch (key)
	{
	case (27): //esc
		glutDestroyWindow(glutGetWindow());
		return; 
	case (87): //W
		mnoznik_predkosci = mnoznik_predkosci / 2;
		return;
	case (119): //w
		mnoznik_predkosci = mnoznik_predkosci / 2;
		return;
	case (115): //s
		mnoznik_predkosci = mnoznik_predkosci * 2;
		return;
	case (83): //S
		mnoznik_predkosci = mnoznik_predkosci * 2;
		return;
	}
}

// obsluga myszy
void mouse(int button, int state, int x, int y)
{
	if (state == GLUT_DOWN)
	{
		mouse_buttons |= 1 << button;
	}
	else if (state == GLUT_UP)
	{
		mouse_buttons = 0;
	}

	mouse_old_x = x;
	mouse_old_y = y;
}

// sterowanie
void motion(int x, int y)
{
	double dx, dy;
	dx = (float)(x - mouse_old_x);
	dy = (float)(y - mouse_old_y);

	if (mouse_buttons & 1)
	{
		rotate_x += dy * 0.2f;
		rotate_y += dx * 0.2f;
	}
	else if (mouse_buttons & 4)
	{
		translate_z += dy * 0.05f;
	}

	mouse_old_x = x;
	mouse_old_y = y;
}

// glut timer
void timerEvent(int value)
{
	if (glutGetWindow())
	{
		glutPostRedisplay();
		glutTimerFunc(REFRESH_DELAY, timerEvent, 0);
	}
}